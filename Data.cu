/*----------
* Authors:
* 	Saúl Contreras (Suulcoder)
* 	Michele Benvenuto
* 	Luis Urbina
* ----------
* Universidad del Valle
* Programación de Microprocesadores
* Semestre 4, 2019
* ----------
*/


#include <hip/hip_runtime.h>
#include <iostream> //cout, cin, cerr
#include <fstream> //file processing
#include <cstdlib> //exit function
#include <string.h>
using namespace std;

#define N 87395 //Number of data in the csv

__global__ void getK(float *a,float *c )
{
	float A = 1000000000.0f;
	float e = 2.71828182846f;
	float Ea = 45000.0f;
	float R = 8.314f;
	int myID = threadIdx.x + blockDim.x * blockIdx.x;				
	if (myID < N)
	{
		c[myID] = (A*float(pow(e,(-Ea)/(R*(a[myID]+273.15f)))));
	}
}

int main(int argc, char** argv)
{

	hipStream_t stream1;							// stream1 and stream2 instantiation
	hipStream_t stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	
	float *a1, *c1; 									// stream 1 mem ptrs
	float *dev_a1, *dev_c1; 						// stream 1 mem ptrs
	
	//stream 1
	hipMalloc( (void**)&dev_a1, N * sizeof(float));									//CudaMalloc
	hipMalloc( (void**)&dev_c1, N * sizeof(float));


	hipHostAlloc( (void**)&a1, N * sizeof(int), hipHostMallocDefault);				//CudaHostAlloc allowing the device to get access to mem. 
	hipHostAlloc( (void**)&c1, N * sizeof(int), hipHostMallocDefault);

	ifstream read("data.csv",ios::in);
	if(!read){
		cerr<<"Fail to read data.csv"<<endl;
	  	exit(EXIT_FAILURE);
	}
	int count = 0;
	string row;
	while(read>>row){
		if(count!=0){
			std::string delimiter = ",";
			if(count%2==0){
				std::string token = row.substr(row.find(delimiter)+1);
				double temp = ::atof(token.c_str());
				float temperature = float(temp);
				a1[count/2] = temperature;
			}			
		}
	  	count++;
	}

	for(int i=0;i<N;i+= N*2) { // loop over data in chunks
	// interweave stream 1 and steam 2
		if(i%2==0){
			hipMemcpyAsync(dev_a1,a1,N*sizeof(int),hipMemcpyHostToDevice,stream1);			//Copy N*Size(int) bytes from a1 to dev_a1, host to device
			getK<<<(int)ceil(N/1024)+1,1024,0,stream1>>>(dev_a1,dev_c1);
			hipMemcpyAsync(c1,dev_c1,N*sizeof(int),hipMemcpyDeviceToHost,stream1);
		}
		else{
			hipMemcpyAsync(dev_a1,a1,N*sizeof(int),hipMemcpyHostToDevice,stream2);			//Copy N*Size(int) bytes from a1 to dev_a1, host to device
			getK<<<(int)ceil(N/1024)+1,1024,0,stream1>>>(dev_a1,dev_c1);
			hipMemcpyAsync(c1,dev_c1,N*sizeof(int),hipMemcpyDeviceToHost,stream2);
		}
	}
	
	for (int k=0;k<N-1;k++){
		cout<<"Dato: "<<k<<" | Value of K: "<<c1[k]<<"\n";
	}

	cout<<"\n\n\n------------------------------Values of K by period:------------------------------";
	cout<<"\n\n          All values returned are based on the Cyclopentadiene Dimerization";
	cout<<"\n\n\n                                H2 + I2 --> 2HI                              \n\n\n";
	std::ofstream myfile;
    myfile.open ("outData.csv");
    myfile<<"Hour,People,Velocity of reaction\n";
	int medPerPeriod = 12000; //300 Data taken per second 12000 in 1 period
	float sum = 0;
	int period = 0;
	for (int k=0;k<N-1;k++){
		sum+=c1[k];
		if(k%medPerPeriod==0&&k!=0){
			period++;
			int people = 0;
			std::string hour = " ";
			if(period==1){
				hour = "07:00 - 07:50";
				people = 38;
			}
			else if(period==2){
				hour = "07:50 - 08:40";
				people = 37;
			}
			else if(period==3){
				hour = "08:40 - 09:30";
				people = 36;
			}
			else if(period==4){
				hour = "09:30 - 10:15";
				people = 36;
			}
			else if(period==5){
				hour = "10:15 - 10:40";
				people = 3;
			}
			else if(period==6){
				hour = "10:40 - 11:30";
				people = 34;
			}
			else if(period==7){
				hour = "11:30 - 12:15";
				people = 35;
			}
			double average = double(sum)/double(medPerPeriod);
			double velocity = (average*0.05*0.05);
			cout<<"\tHour: "<<hour<<"\tPeople: "<<people<<"\tVelocity of reaction: "<< velocity<<"s\n";
			myfile<<hour<<","<<people<<","<< velocity<<"s\n";
			sum=0;
		}
	}
	cout<<"-----------------------------------------------------------------------------------\n\n\n";
	myfile.close();
	hipStreamDestroy(stream1);					//Destruir cudaStreamDestroy(stream1)
	return 0;
}