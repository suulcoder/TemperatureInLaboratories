/*----------
* Authors:
* 	Saúl Contreras (Suulcoder)
* 	Michele Benvenuto
* 	Luis Urbina
* ----------
* Universidad del Valle
* Programación de Microprocesadores
* Semestre 4, 2019
* ----------
*/

#include <iostream> //cout, cin, cerr
#include <unistd.h>
#include <cmath>
#include <fstream> //file processing
#include <cstdlib> //exit function
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
using namespace std;

#define N 87395 //Number of data in the csv

__global__ void getK(float *a,float *c )
{
	float A = 156.15f;
	float e = 2.71828182846f;
	float Ea = 23.515f;
	float R = 8.314f;
	int myID = threadIdx.x + blockDim.x * blockIdx.x;				
	if (myID < N)
	{
		c[myID] = (A*float(pow(e,(-Ea)/(R*a[myID]))));
	}
}

int main(int argc, char** argv)
{

	hipStream_t stream1;							// strem2 instantiation
	hipStreamCreate(&stream1);
	
	float *a1, *c1; 									// stream 1 mem ptrs
	float *dev_a1, *dev_c1; 						// stream 1 mem ptrs
	
	//stream 1
	hipMalloc( (void**)&dev_a1, N * sizeof(float));									//CudaMalloc
	hipMalloc( (void**)&dev_c1, N * sizeof(float));

	hipHostAlloc( (void**)&a1, N * sizeof(int), hipHostMallocDefault);				//CudaHostAlloc allowing the device to get access to mem. 
	hipHostAlloc( (void**)&c1, N * sizeof(int), hipHostMallocDefault);
	ifstream read("data.csv",ios::in);
	if(!read){
		cerr<<"Fail to read data.csv"<<endl;
	  	exit(EXIT_FAILURE);
	}
	int count = 0;
	string row;
	while(read>>row){
		if(count!=0){
			std::string delimiter = ",";
			std::string token = row.substr(row.find(delimiter)+1);
			double temp = ::atof(token.c_str());
			float temperature = float(temp);
			a1[count] = temperature;
		}
	  	count++;
	}

	for(int i=0;i < N;i+= N*2) { // loop over data in chunks
	// interweave stream 1 and steam 2
		hipMemcpyAsync(dev_a1,a1,N*sizeof(int),hipMemcpyHostToDevice,stream1);			//Copy N*Size(int) bytes from a1 to dev_a1, host to device
		getK<<<(int)ceil(N/1024)+1,1024,0,stream1>>>(dev_a1,dev_c1);
		hipMemcpyAsync(c1,dev_c1,N*sizeof(int),hipMemcpyDeviceToHost,stream1);
	}

	for (int k=0;k<N;k++){
		cout<<c1[k]<<"\n";
	}

	hipStreamDestroy(stream1);					//Destruir cudaStreamDestroy(stream1)
	return 0;
}